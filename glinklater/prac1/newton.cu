#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////
// GPU version of Newton's method to calculate square roots
// Inefficient code to calculate square root of n random numbers using CURAND
////////////////////////////////////////////////////////////////////////

#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hip/hip_runtime_api.h>

using namespace std;
#define MAXVAL 100000   // values must be between 1 and MAXVAL - so there will be duplicates

__global__ void squareRoot ( int n, unsigned int *x, float *y, int iter )
// Applies Newton's method to compute the square root 
// of the appropriate int in x and places result in y
{
   int i = blockIdx.x*blockDim.x + threadIdx.x;
   float inc;
   int c = (abs((int) x[i]) % MAXVAL) + 1; // change random number generated to a value between 1 and MAXVAL
   float r = c;
   if (i<n)   // to prevent excess processors from accessible out-of-bounds data
   { for(int j=0; j<iter; j++)   // iterations of Newton's method - based on parameter iter
        { // actual Newton's method
          inc = r + r;
          inc = (r*r - c)/inc;
          r = r - inc;
        }
	   x[i] = c;  // copy actual number used into x
       y[i] = r;  // write in result - i.e. square root of x value
   }
}


int main (int argc, char*argv[] )
{     int iterations = 10; // number of iterations for Newton's method
      int n = 1000000; //  dimension of data
      int block = 32; // block size
	  int grid = n / block + 1;  // round up number of blocks

 // initialise card

	findCudaDevice(argc, (const char**) argv);   // from hip/hip_runtime_api.h 
 
  // initialise CUDA timing

	float milli;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// allocate memory on host and device
		unsigned int *h_x = (unsigned int *)malloc(sizeof(unsigned int)*n);     
		float *h_y = (float *)malloc(sizeof(float)*n);     
		unsigned int *d_x;  // memory for initial numbers
		checkCudaErrors(hipMalloc((void**)&d_x,sizeof(unsigned int)*n));
		float *d_y;  // memory for result
		checkCudaErrors(hipMalloc((void**)&d_y,sizeof(float)*n));
    
	// random number generation
		hipEventRecord(start);  // start timing

		hiprandGenerator_t gen;
		checkCudaErrors( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
		checkCudaErrors( hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL) );
		checkCudaErrors( hiprandGenerate(gen, d_x, n) );
 
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milli, start, stop);  // time random generation

		printf("CURAND normal RNG  execution time (ms): %f,  samples/sec: %e \n",
          milli, n/(0.001*milli));

	// execute kernel and time it

		hipEventRecord(start); // start timing

		squareRoot<<<grid,block>>>(n, d_x, d_y, iterations);

		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milli, start, stop);  // stop timing actual kernel execution

		printf("Square root kernel execution time (ms): %f \n",milli);

		checkCudaErrors(hipDeviceSynchronize());  // flush print queues

		// copy back results
		checkCudaErrors( hipMemcpy(h_y, d_y, sizeof(float)*n, hipMemcpyDeviceToHost) );	
		checkCudaErrors( hipMemcpy(h_x, d_x, sizeof(unsigned int)*n, hipMemcpyDeviceToHost) );	  

  /*    for (int k = 0; k < n; k++) // Really just for testing to check how close results are to actual square roots
      {
         printf("x = %u", h_x[k]);
         printf("  sqrt(x) = %f", h_y[k]);
         float z = h_y[k]*h_y[k];
         printf("  diff = %f\n", z - h_x[k]);
      }  */

	 // Tidy up library

		checkCudaErrors( hiprandDestroyGenerator(gen) );

	// Release memory and exit cleanly

		free(h_x);
		free(h_y);
		checkCudaErrors( hipFree(d_x) );
		checkCudaErrors( hipFree(d_y) );

	// CUDA exit 

		hipDeviceReset();
}

