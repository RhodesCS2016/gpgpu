#include "hip/hip_runtime.h"

#include <ctype.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <assert.h>
#include <utility>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <queue>
#include "hip/hip_runtime_api.h"
#include "mylib.h"

typedef std::queue<hipStream_t*> StreamQueue;

__global__ void simpleLifeKernel(const ubyte *lifeData, uint worldWidth, uint worldSize, ubyte *resultLifeData) {
  for (uint cellId = __mul24(blockIdx.x, blockDim.x) + threadIdx.x; cellId < worldSize; cellId += blockDim.x * gridDim.x) {
    uint x = cellId % worldWidth;
    uint yAbs = cellId - x;
    uint xLeft = (x + worldWidth - 1) % worldWidth;
    uint xRight = (x + 1) % worldWidth;
    uint yAbsUp = (yAbs + worldSize - worldWidth) % worldSize;
    uint yAbsDown = (yAbs + worldWidth) % worldSize;

    uint aliveCells = lifeData[xLeft + yAbsUp] + lifeData[x + yAbsUp]
      + lifeData[xRight + yAbsUp] + lifeData[xLeft + yAbs] + lifeData[xRight + yAbs]
      + lifeData[xLeft + yAbsDown] + lifeData[x + yAbsDown] + lifeData[xRight + yAbsDown];

    resultLifeData[x + yAbs] =
      aliveCells == 3 || (aliveCells == 2 && lifeData[x + yAbs]) ? 1 : 0;
  }
}

// populate a queue with specified number of non-blocking streams.
void setupCUDAStreams(StreamQueue *stream_q, int stream_count) {
  for (int i = 0; i < stream_count; i++) {
    hipStream_t tmp;
    hipStreamCreateWithFlags(&tmp, hipStreamNonBlocking);
    stream_q->push(&tmp);
  }
}

// get the next stream from the queue (round-robin)
hipStream_t *getNextCUDAStream(StreamQueue *stream_q) {
  hipStream_t *next = stream_q->front();
  stream_q->pop();
  stream_q->push(next);
  return next;
}

void runSimpleLifeKernel(
  ubyte *d_lifeData,
  ubyte *d_lifeDataBuffer,
  size_t worldWidth,
  size_t worldHeight,
  size_t iterationsCount,
  ushort threadsCount,
  StreamQueue *stream_q)
{
  assert((worldWidth * worldHeight) % threadsCount == 0);
  size_t reqBlocksCount = (worldWidth * worldHeight) / threadsCount;
  ushort blocksCount = (ushort)std::min((size_t)32768, reqBlocksCount);

  for (size_t i = 0; i < iterationsCount; ++i) {
    simpleLifeKernel<<<blocksCount, threadsCount, 0, *getNextCUDAStream(stream_q)>>>(d_lifeData, worldWidth, worldWidth*worldHeight, d_lifeDataBuffer);
    std::swap(d_lifeData, d_lifeDataBuffer);
  }
}

world *gameWorld;
board *gameBoard;
FILE *out_file;

int main (int argc, char **argv)
{
  int iterations = 10000;
  ushort threadsCount = 64;
  char *in_filename = NULL;
  char *out_filename = NULL;
  int stream_count = 1;
  size_t board_size = 48;

  float milli;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

  gameBoard = (board*)malloc(sizeof(board));

  opterr = 0;
  int c;

  while ((c = getopt (argc, argv, (const char*)"o:f:i:t:s:x:")) != -1) {
    switch (c) {
      case 'i':
        iterations = atoi(optarg);
        break;
      case 'f':
        in_filename = optarg;
        break;
      case 'o':
        out_filename = optarg;
        break;
      case 't':
        threadsCount = atoi(optarg);
        break;
      case 's':
        board_size = atoi(optarg);
        break;
      case 'x':
        stream_count = atoi(optarg);
        break;
      case '?':
        break;
      default:
        break;
    }
  }

  printf("iterations: %d\n", iterations);
  printf("in_file: %s\n", in_filename);
  printf("out_file: %s\n", out_filename);
  printf("threadsCount: %u\n", threadsCount);
  printf("\n");

  if (!in_filename) {
    printf("Please specify a board file\n");
    exit(1);
  }

  initWorld(board_size, board_size, gameWorld);
  initBoard(fopen(in_filename, "r"), gameBoard, gameWorld);
  if (out_filename) out_file = fopen(out_filename, "w+");

  ubyte *d_data;
  ubyte *d_resultData;
  StreamQueue stream_q;
  setupCUDAStreams(&stream_q, stream_count);

  checkCudaErrors(hipMalloc((ubyte**)&d_data, gameWorld->dataLength));
  checkCudaErrors(hipMemset(d_data, 0, gameWorld->dataLength));
  checkCudaErrors(hipMemcpy(d_data, gameBoard->data, gameWorld->dataLength, hipMemcpyHostToDevice));

  checkCudaErrors(hipMalloc((ubyte**)&d_resultData, gameWorld->dataLength));
  checkCudaErrors(hipMemset(d_resultData, 0, gameWorld->dataLength));

  hipEventRecord(start);  // start timing
  runSimpleLifeKernel(d_data, d_resultData, board_size, board_size, iterations, threadsCount, &stream_q);
  hipEventRecord(stop);  // stop timing
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);  //

  printf("Elapsed Time for %d iterations: %f", iterations, milli);

  checkCudaErrors(hipMemcpy(gameBoard->data, d_data, gameWorld->dataLength, hipMemcpyDeviceToHost));
  printBoard(gameBoard->data, gameWorld);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  free(gameBoard->data);
  free(gameBoard->resultData);
  if (out_filename) fclose(out_file);
  printf("\n");
  checkCudaErrors(hipDeviceReset());
}
