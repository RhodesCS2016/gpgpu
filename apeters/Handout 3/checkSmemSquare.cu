#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hip/hip_runtime_api.h"
/* Example code taken from "Cheng J. et al. Professional CUDA C Programming"
 * An example of using shared memory to transpose square thread coordinates
 * of a CUDA grid into a global memory array. Different kernels below
 * demonstrate performing reads and writes with different ordering, as well as
 * optimizing using memory padding.
 */

#define BDIMX 32
#define BDIMY 64
#define IPAD  1

void printData(char *msg, int *in,  const int size)
{
    printf("%s: ", msg);

    for (int i = 0; i < size; i++)
    {
        printf("%5d", in[i]);
        fflush(stdout);
    }

    printf("\n");
    return;
}

__global__ void setRowReadRow (int *out)
{
    // static shared memory
    __shared__ int tile[BDIMY][BDIMX];

    // mapping from thread index to global memory index
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    // shared memory store operation
    tile[threadIdx.y][threadIdx.x] = idx;

    // wait for all threads to complete
    __syncthreads();

    // shared memory load operation
    out[idx] = tile[threadIdx.y][threadIdx.x] ;
}

__global__ void setColReadCol (int *out)
{
    // static shared memory
    __shared__ int tile[BDIMX][BDIMY];

    // mapping from thread index to global memory index
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    // shared memory store operation
    tile[threadIdx.x][threadIdx.y] = idx;

    // wait for all threads to complete
    __syncthreads();

    // shared memory load operation
    out[idx] = tile[threadIdx.x][threadIdx.y];
}

__global__ void setRowReadCol(int *out)
{
    // static shared memory
    __shared__ int tile[BDIMY][BDIMX];

    // mapping from thread index to global memory index
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    // shared memory store operation
    tile[threadIdx.y][threadIdx.x] = idx;

    // wait for all threads to complete
    __syncthreads();

    // shared memory load operation
    out[idx] = tile[threadIdx.x][threadIdx.y];
}


__global__ void setRowReadColDyn(int *out)
{
    // dynamic shared memory
    extern  __shared__ int tile[];

    // mapping from thread index to global memory index
    unsigned int row_idx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int col_idx = threadIdx.x * blockDim.y + threadIdx.y;

    // shared memory store operation
    tile[row_idx] = row_idx;

    // wait for all threads to complete
    __syncthreads();

    // shared memory load operation
    out[row_idx] = tile[col_idx];
}

__global__ void setRowReadColPad(int *out)
{
    // static shared memory
    __shared__ int tile[BDIMY][BDIMX + IPAD];

    // mapping from thread index to global memory offset
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    // shared memory store operation
    tile[threadIdx.y][threadIdx.x] = idx;

    // wait for all threads to complete
    __syncthreads();

    // shared memory load operation
    out[idx] = tile[threadIdx.x][threadIdx.y];
}


int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    checkCudaErrors(hipSetDevice(dev));

    hipSharedMemConfig pConfig;
    checkCudaErrors(hipDeviceGetSharedMemConfig ( &pConfig ));
    printf("with Bank Mode:%s ", pConfig == 1 ? "4-Byte" : "8-Byte");

    // set up array size 2048
    int nx = BDIMX;
    int ny = BDIMY;

    bool iprintf = 0;

    if (argc > 1) iprintf = atoi(argv[1]);

    size_t nBytes = nx * ny * sizeof(int);

    // execution configuration
    dim3 block (BDIMX, BDIMY);
    dim3 grid  (1, 1);
    printf("<<< grid (%d,%d) block (%d,%d)>>>\n", grid.x, grid.y, block.x,
           block.y);

    // allocate device memory
    int *d_C;
    checkCudaErrors(hipMalloc((int**)&d_C, nBytes));
    int *gpuRef  = (int *)malloc(nBytes);

    checkCudaErrors(hipMemset(d_C, 0, nBytes));
    setColReadCol<<<grid, block>>>(d_C);
    checkCudaErrors(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprintf)  printData("set col read col   ", gpuRef, nx * ny);

    checkCudaErrors(hipMemset(d_C, 0, nBytes));
    setRowReadRow<<<grid, block>>>(d_C);
    checkCudaErrors(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprintf)  printData("set row read row   ", gpuRef, nx * ny);

    checkCudaErrors(hipMemset(d_C, 0, nBytes));
    setRowReadCol<<<grid, block>>>(d_C);
    checkCudaErrors(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprintf)  printData("set row read col   ", gpuRef, nx * ny);

    checkCudaErrors(hipMemset(d_C, 0, nBytes));
    setRowReadColDyn<<<grid, block, BDIMX*BDIMY*sizeof(int)>>>(d_C);
    checkCudaErrors(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprintf)  printData("set row read col dyn", gpuRef, nx * ny);

    checkCudaErrors(hipMemset(d_C, 0, nBytes));
    setRowReadColPad<<<grid, block>>>(d_C);
    checkCudaErrors(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprintf)  printData("set row read col pad", gpuRef, nx * ny);

    // free host and device memory
    checkCudaErrors(hipFree(d_C));
    free(gpuRef);

    // reset device
    checkCudaErrors(hipDeviceReset());
    return EXIT_SUCCESS;
}